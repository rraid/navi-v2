
#include <hip/hip_runtime.h>
#ifdef __NVCC__

#include <cuda.h>
#include <cuda_runtime.h>
#include <cuda_runtime_api.h>

#define IJ2C(i,j,ld)(((j)*(ld))+(i))
#define checkCudaErrors(val) __check( (val), #val, __FILE__, __LINE__)
template<typename T>
void __check(T err, const char* const func, const char* const file, const int line) {
  if (err != cudaSuccess) {
    std::cerr << "CUDA error at: " << file << ":" << line << std::endl;
    std::cerr << cudaGetErrorString(err) << " " << func << std::endl;
    exit(1);
  }
}

template<typename T>
T *zeros(int rows, int cols) {
  uint8_t *dbuf;
  int n_elem = rows * cols;
  checkCudaErrors(cudaMalloc(&dbuf, sizeof(T) * n_elem));
  checkCudaErrors(cudaMemset(dbuf, 0, sizeof(T) * n_elem));
  return dbuf;
}

void compute(void *xPotential, void *yPotential, void *gridmap,
    int rows, int cols, int x, int y) {
  uint8_t *closed = zeros<uint8_t>(rows, cols);
  uint32_t *opened = gheap();
  uint32_t *openedLength = zeros<uint32_t>(1, 1);
  float *gScore = zeros<float>(rows, cols);
  float *dxPotential = zeros<float>(rows, cols);
  float *dyPotential = zeros<float>(rows, cols);
  float *c_space = zeros<float>(rows, cols);

  uint8_t *action_map = zeros<float>(256, 4);

  uint32_t openlen = 1;
  while (openlen > 0) {
    dim3 blocksize(256, 1, 1);
    dim3 gridSize((openlen-1)/256+1, 1, 1);
    gpu_inplace_sort<<<gridSize, blockSize>>>(opened, openedLength);
    checkCudaErrors(cudaGetLastError());
    
    gpu_close_min<<<gridSize, blockSize>>>(opened, openedLength, closed);
    checkCudaErrors(cudaGetLastError());

    gpu_map_actions<<<gridSize, blockSize>>>(opened, openedLength, action_map);
    checkCudaErrors(cudaGetLastError());

    // TODO: finish when not so tired
  }
}

#else

void compute(void *xPotential, void *yPotential, void *gridmap,
    int rows, int cols, int x, int y) {
  printf("Cannot compute path - no CUDA library found\n");
}

#endif
